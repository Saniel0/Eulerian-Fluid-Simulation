#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "solver_cuda.h"

__device__ __host__ inline int coord(int w, int h, int width) {
    return ((h * width) + w);
}

Solver_cuda::Solver_cuda(int w, int h, int iters, float relax_factor, float dt) {
    this->width = w + 2; // +2 for border
    this->height = h + 2;
    this->iters = iters;
    this->relax_factor = relax_factor;
    this->dt = dt;
    // allocate memory on GPU (device)
    hipMalloc(&grid_s, width * height * sizeof(uint8_t));
    hipMalloc(&grid_u, width * height * sizeof(float));
    hipMalloc(&tmp_u, width * height * sizeof(float));
    hipMalloc(&grid_v, width * height * sizeof(float));
    hipMalloc(&tmp_v, width * height * sizeof(float));
    hipMalloc(&grid_m, width * height * sizeof(float));
    hipMalloc(&tmp_m, width * height * sizeof(float));
    hipMalloc(&frame_data, w * h * sizeof(float));
    // allocate memory on CPU (host)
    this->frame_data_host = new uint8_t[w * h];
}

Solver_cuda::~Solver_cuda() {
    hipFree(grid_s);
    hipFree(grid_u);
    hipFree(tmp_u);
    hipFree(grid_v);
    hipFree(tmp_v);
    hipFree(grid_m);
    hipFree(tmp_m);
    delete[] frame_data_host;
}

__global__ void iterate_compression_kernel(uint8_t *grid_s, float *grid_u, float *grid_v, int width, int height, float relax_factor, int iteration) {
    int w = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int h = blockIdx.y * blockDim.y + threadIdx.y + 1;
    // check bounds
    if (w >= width - 1 || h >= height - 1) return;
    // check if we are on correct cell
    if ((w + h) % 2 == iteration % 2) return;

    if (grid_s[h * width + w] == 0) return;

    int s_up = grid_s[(h-1) * width + w];
    int s_down = grid_s[(h+1) * width + w];
    int s_left = grid_s[h * width + w - 1];
    int s_right = grid_s[h * width + w + 1];
    int s = s_up + s_down + s_left + s_right;

    float div = - grid_v[h * width + w] + grid_v[(h+1) * width + w] - grid_u[h * width + w] + grid_u[h * width + w + 1];
    float p = div / s;
    p *= relax_factor;

    grid_v[h * width + w] += s_up * p;
    grid_v[(h+1) * width + w] -= s_down * p;
    grid_u[h * width + w] += s_left * p;
    grid_u[h * width + w + 1] -= s_right * p;
}

void Solver_cuda::iterate_compression() {
    // Define block and grid sizes
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    // Run the iterations with CUDA
    for (int i = 0; i < iters * 2; ++i) {
        iterate_compression_kernel<<<gridDim, blockDim>>>(grid_s, grid_u, grid_v, width, height, relax_factor, i);
        hipDeviceSynchronize(); // Ensure all threads complete before the next iteration
    }
}

__global__ void advect_velocities_kernel(uint8_t *grid_s, float *grid_u, float *grid_v, float *tmp_u, float *tmp_v, int width, int height, float dt) {
    int w = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int h = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (w >= width - 1 || h >= height - 1) return;
    // if there is collision object
    if (grid_s[coord(w, h, width)] == 0) return;
    tmp_v[coord(w, h, width)] = grid_v[coord(w, h, width)];
    tmp_u[coord(w, h, width)] = grid_u[coord(w, h, width)];
    if (grid_s[coord(w, h-1, width)] != 0) {
        float v = grid_v[coord(w, h, width)];
        // get u part of the vector by avareging 4 surrounding 'u's
        float u = (grid_u[coord(w, h-1, width)] + grid_u[coord(w+1, h-1, width)] + grid_u[coord(w, h, width)] + grid_u[coord(w+1, h, width)]) / 4;
        // backtrack based on timestep
        float x = (float) w - (u * dt);
        float y = (float) h - (v * dt);
        // correct if backtracked coordinates are out of bounds
        x = max(min(x, (float) (width-1)), (float) 1.0);
        y = max(min(y, (float) (height-1)), (float) 1.0);
        if (grid_s[coord((int) x, (int) y, width)] != 0) {
            int idx = (int) x;
            int idy = (int) y;
            float ratio_x = x - idx;
            float ratio_y = y - idy;
            float tmp1 = (grid_v[coord(idx, idy, width)] * (1.0 - ratio_x)) + (grid_v[coord(idx+1, idy, width)] * ratio_x);
            float tmp2 = (grid_v[coord(idx, idy+1, width)] * (1.0 - ratio_x)) + (grid_v[coord(idx+1, idy+1, width)] * ratio_x);
            float tmp = (tmp1 * (1.0 - ratio_y)) + (tmp2 * ratio_y);
            tmp_v[coord(w, h, width)] = tmp;
        }
    }
    // if there is not obstacle leftwards
    if (grid_s[coord(w-1, h, width)] != 0) {
        float u = grid_u[coord(w, h, width)];
        // get v part of the vector by avareging 4 surrounding 'v's
        float v = (grid_v[coord(w-1, h, width)] + grid_v[coord(w, h, width)] + grid_v[coord(w-1, h+1, width)] + grid_v[coord(w, h+1, width)]) / 4;
        // backtrack based on timestep
        float x = (float) w - (u * dt);
        float y = (float) h - (v * dt);
        // correct if backtracked coordinates are out of bounds
        x = max(min(x, (float) (width-1)), (float) 1.0);
        y = max(min(y, (float) (height-1)), (float) 1.0);
        if (grid_s[coord((int) x, (int) y, width)] != 0) {
            int idx = (int) x;
            int idy = (int) y;
            float ratio_x = x - idx;
            float ratio_y = y - idy;
            float tmp1 = (grid_u[coord(idx, idy, width)] * (1.0 - ratio_x)) + (grid_u[coord(idx+1, idy, width)] * ratio_x);
            float tmp2 = (grid_u[coord(idx, idy+1, width)] * (1.0 - ratio_x)) + (grid_u[coord(idx+1, idy+1, width)] * ratio_x);
            float tmp = (tmp1 * (1.0 - ratio_y)) + (tmp2 * ratio_y);
            tmp_u[coord(w, h, width)] = tmp;
        }
    }
}

void Solver_cuda::advect_velocities() {
    // Define block and grid sizes
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    advect_velocities_kernel<<<gridDim, blockDim>>>(grid_s, grid_u, grid_v, tmp_u, tmp_v, width, height, dt);
    hipDeviceSynchronize(); // Ensure all threads complete
    // swap the tmp and main grids
    float *tmp_ptr = grid_v;
    grid_v = tmp_v;
    tmp_v = tmp_ptr;
    tmp_ptr = grid_u;
    grid_u = tmp_u;
    tmp_u = tmp_ptr;
}

__global__ void advect_smoke_kernel(uint8_t *grid_s, float *grid_u, float *grid_v, float *grid_m, float *tmp_m, int width, int height, float dt) {
    int w = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int h = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (w >= width - 1 || h >= height - 1) return;
    // if there is collision object
    if (grid_s[coord(w, h, width)] == 0) return;
    tmp_m[coord(w, h, width)] = grid_m[coord(w, h, width)];
    float v = (grid_v[coord(w, h, width)] + grid_v[coord(w, h+1, width)]) / 2;  
    float u = (grid_u[coord(w, h, width)] + grid_u[coord(w+1, h, width)]) / 2;
    float x = (float) w - (u * dt);
    float y = (float) h - (v * dt);
    // correct if backtracked coordinates are out of bounds
    x = max(min(x, (float) (width-1)), (float) 0.0);
    y = max(min(y, (float) (height-1)), (float) 0.0);
    if (grid_s[coord((int) x, (int) y, width)] != 0 || ((int) x) == 0) {
        int idx = (int) x;
        int idy = (int) y;
        float ratio_x = x - idx;
        float ratio_y = y - idy;
        float tmp1 = (grid_m[coord(idx, idy, width)] * (1.0 - ratio_x)) + (grid_m[coord(idx+1, idy, width)] * ratio_x);
        float tmp2 = (grid_m[coord(idx, idy+1, width)] * (1.0 - ratio_x)) + (grid_m[coord(idx+1, idy+1, width)] * ratio_x);
        float tmp = (tmp1 * (1.0 - ratio_y)) + (tmp2 * ratio_y);
        tmp_m[coord(w, h, width)] = tmp;
    }
}

void Solver_cuda::advect_smoke() {
    // Define block and grid sizes
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    advect_smoke_kernel<<<gridDim, blockDim>>>(grid_s, grid_u, grid_v, grid_m, tmp_m, width, height, dt);
    hipDeviceSynchronize(); // Ensure all threads complete
    // swap the tmp and main grids
    float *tmp_ptr = grid_m;
    grid_m = tmp_m;
    tmp_m = tmp_ptr;
}

void Solver_cuda::wind_tunnel() {
    float *grid_vC = new float[width*height];
    float *tmp_vC = new float[width*height];
    float *grid_uC = new float[width*height];
    float *tmp_uC = new float[width*height];
    float *grid_mC = new float[width*height];
    float *tmp_mC = new float[width*height];
    uint8_t *grid_sC = new uint8_t[width*height];
    
    for (int i = 0; i < width * height; ++i) {
        grid_vC[i] = 0;
        tmp_vC[i] = 0;
        grid_uC[i] = 16.0;
        tmp_uC[i] = 16.0;
        grid_mC[i] = 0;
        tmp_mC[i] = 0;
    }

    for (int i = (height / 2) - (height / 16); i < (height / 2) + (height / 16); ++i) {
        grid_mC[coord(0, i, width)] = 1.0;
        grid_mC[coord(1, i, width)] = 1.0;
        tmp_mC[coord(0, i, width)] = 1.0;
        tmp_mC[coord(1, i, width)] = 1.0;
    }

    for (int h = 0; h < height; ++h) {
        for (int w = 0; w < width; ++w) {
            // if outside boundary
            if (w == 0 || w == width - 1 || h == 0 || h == height - 1) {
                grid_sC[coord(w, h, width)] = 0;
            }
            // if object inside simulation space
            else if (sqrt((w + 1 - ((height / 2) + (height / 8)))*(w + 1 - ((height / 2) + (height / 8))) + (h + 1 - (width / 4))*(h + 1 - (width / 4))) <= (height / 8) + (height / 16)) {
                grid_sC[coord(w, h, width)] = 0;
                grid_uC[coord(w, h, width)] = 0;
                grid_uC[coord(w+1, h, width)] = 0;
                tmp_uC[coord(w, h, width)] = 0;
                tmp_uC[coord(w+1, h, width)] = 0;
            }
            // if space where fluid can flow
            else {
                grid_sC[coord(w, h, width)] = 1;
            }
        }
    }

    hipMemcpy(grid_s, grid_sC, width * height * sizeof(uint8_t), hipMemcpyHostToDevice);
    hipMemcpy(grid_v, grid_vC, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tmp_v, tmp_vC, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(grid_u, grid_uC, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tmp_u, tmp_uC, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(grid_m, grid_mC, width * height * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(tmp_m, tmp_mC, width * height * sizeof(float), hipMemcpyHostToDevice);
}

__global__ void get_frame_kernel(uint8_t *grid_s, uint8_t *frame_data, float *grid_m, int width, int height) {
    int w = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int h = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (w >= width - 1 || w == 0 || h == 0 || h >= height - 1) return;

    int idx = (h-1)*(width-2) + (w-1);
    frame_data[idx] = 255 - (128 * grid_m[coord(w, h, width)]);
    frame_data[idx] *= grid_s[coord(w, h, width)];
}

uint8_t *Solver_cuda::get_frame() {
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    get_frame_kernel<<<gridDim, blockDim>>>(grid_s, frame_data, grid_m, width, height);
    hipDeviceSynchronize();
    hipMemcpy(frame_data_host, frame_data, (width-2) * (height-2) * sizeof(uint8_t), hipMemcpyDeviceToHost);
    return frame_data_host;
}
